#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


#include "hip/hip_runtime.h"
#include ""


#define n 0.0002
#define p 0.5
#define G 0.75

#define SIZE 1024
#define NUMBER_OF_ITERATIONS 100
#define DEBUG 0


__device__ int idx(int i, int j){
    return (SIZE * i + j);
}


__global__ void foo(double *u1){
    int i = blockIdx.x;  
    int j = threadIdx.x;
    printf("u1[%d,%d]: %.3lf  \t",i, j,  u1[idx(i,j)] );
    
}

__global__ void updateElement(double *u, double *u1, double *u2)
{
    int i = blockIdx.x;  
    int j = threadIdx.x;

    //printf("i: %d j: %d \n", i, j);

    for(int j=0; j < SIZE; j++)
    //for(int i=0; i < SIZE; i++)
    {
        //taken care of by other threads
        if(i == 0 || j == 0 || i == SIZE-1 || j == SIZE-1){
            continue;
        }

        u[idx(i, j)]=  p * 
                                (u1[idx(i-1,j)] + u1[idx(i+1,j)] 
                                +u1[idx(i,j-1)] + u1[idx(i,j+1)] 
                            - 4 * u1[idx(i, j)])  
                            + 2 * u1[idx(i, j)] - (1-n) * u2[idx(i, j)];

        if(j==1){
            u[idx(i,0)] = G * u[idx(i, j)];

            //top left corner
            if(i == 1){
                u[idx(0,0)] = G * u[idx(1,0)];
            }

            //top right corner
            if(i == SIZE-2){
                u[idx(SIZE-1,0)] = G * u[idx(SIZE-2, 0)];
            }

        }

        if(i==1){
            u[idx(0, j)] = G * u[idx(i, j)];
            //bottom left corner
            if(j==SIZE-2){
                u[idx(0,SIZE-1)] = G * u[idx(0, SIZE-2)];
            }
        }

        if(j == SIZE-2){
            u[idx(i, SIZE-1)]  = G * u[idx(i, j)];
        }

        if(i == SIZE-2){
            u[idx(SIZE-1, j)]  = G * u[idx(i, j)];
            //bottom right corner
            if(j== SIZE-2){
                u[idx(SIZE-1, SIZE-1)] = G * u[idx(SIZE-1, SIZE-2)];
            }
        }
    }
}


void printMatrix(double* u){
        printf("\n");
        for(int i = 0; i < SIZE * SIZE; i++){
            printf("%.3lf", u[i]);
            printf("\t");
            if((i+1) %  SIZE == 0 && i > 0){
                printf("\n");
            }
        }
}


int main(){

    double* u  = static_cast<double*>(malloc(sizeof(double) * SIZE * SIZE ));
    double* u1 = static_cast<double*>(malloc(sizeof(double) * SIZE * SIZE ));
    double* u2 = static_cast<double*>(malloc(sizeof(double) * SIZE * SIZE ));


    //initialize to 0
    for(int i = 0; i < SIZE * SIZE; i++){
        //u[i] = 0;
        u1[i] = 0;
        u2[i] = 0;
    }

    //hit that drummmm
    //u1[idx(SIZE/2, SIZE/2)] = 1;
    u1[(SIZE * SIZE/2 + SIZE/2)] = 1;
    //printMatrix(u1);

    clock_t start, end;
    double cpu_time_used;
    
    double* u_dev, *u2_dev; 
    double *u1_dev;
    hipMalloc((void **)&u_dev, SIZE*SIZE *sizeof(double));
    hipMalloc((void **)&u1_dev, SIZE*SIZE *sizeof(double));
    hipMalloc((void **)&u2_dev, SIZE*SIZE *sizeof(double));  
  
    hipMemcpy(u_dev, u, SIZE*SIZE *sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(u1_dev, u1, SIZE*SIZE *sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(u2_dev, u2, SIZE*SIZE *sizeof(double), hipMemcpyHostToDevice);

    u1[(SIZE * SIZE/2 + SIZE/2)] = 1;

    start = clock();

    for(int i = 0; i < NUMBER_OF_ITERATIONS ; i++){
        updateElement << <SIZE, 1 >> > (u_dev, u1_dev, u2_dev);
        hipDeviceSynchronize();
        
        if(DEBUG){
            hipMemcpy(u, u_dev, SIZE*SIZE *sizeof(double), hipMemcpyDeviceToHost);
            //hipMemcpy(u1, u1_dev, SIZE*SIZE *sizeof(double), hipMemcpyDeviceToHost);

            //printMatrix(u);
            printf("\n\n%lf", u[(SIZE * SIZE/2 + SIZE/2)] );
        }
        hipMemcpy(u2_dev, u1_dev, SIZE*SIZE *sizeof(double), hipMemcpyDeviceToDevice);
        hipMemcpy(u1_dev, u_dev, SIZE*SIZE *sizeof(double), hipMemcpyDeviceToDevice);
    }
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("\n Part 3_many blocks, time: \t%lf \n", cpu_time_used);
    hipFree(u_dev);
    hipFree(u1_dev);
    hipFree(u2_dev);
    free(u);
    free(u1);
    free(u2);
}